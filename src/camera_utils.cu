#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "camera_utils.cuh"
#include "stb_image.h"
#include <cmath>
#include <eigen3/Eigen/Dense>
#include <filesystem>
#include <iostream>

torch::Tensor getWorld2View2(const Eigen::Matrix3d& R, const Eigen::Vector3d& t,
                             const Eigen::Vector3d& translate /*= Eigen::Vector3d::Zero()*/, float scale /*= 1.0*/) {
    Eigen::Matrix4d Rt = Eigen::Matrix4d::Zero();
    Rt.block<3, 3>(0, 0) = R.transpose();
    Rt.block<3, 1>(0, 3) = t;
    Rt(3, 3) = 1.0;

    Eigen::Matrix4d C2W = Rt.inverse();
    Eigen::Vector3d cam_center = C2W.block<3, 1>(0, 3);
    cam_center = (cam_center + translate) * scale;
    C2W.block<3, 1>(0, 3) = cam_center;
    Rt = C2W.inverse();
    // Here we create a torch::Tensor from the Eigen::Matrix
    // Note that the tensor will be on the CPU, you may want to move it to the desired device later
    auto RtTensor = torch::from_blob(Rt.data(), {4, 4}, torch::kFloat32);

    // clone the tensor to allocate new memory, as from_blob shares the same memory
    // this step is important if Rt will go out of scope and the tensor will be used later
    return RtTensor.clone();
}

Eigen::Matrix4d getWorld2View2Eigen(const Eigen::Matrix3d& R, const Eigen::Vector3d& t,
                                    const Eigen::Vector3d& translate /*= Eigen::Vector3d::Zero()*/, float scale /*= 1.0*/) {
    Eigen::Matrix4d Rt = Eigen::Matrix4d::Zero();
    Rt.block<3, 3>(0, 0) = R.transpose();
    Rt.block<3, 1>(0, 3) = t;
    Rt(3, 3) = 1.0;

    Eigen::Matrix4d C2W = Rt.inverse();
    Eigen::Vector3d cam_center = C2W.block<3, 1>(0, 3);
    cam_center = (cam_center + translate) * scale;
    C2W.block<3, 1>(0, 3) = cam_center;
    Rt = C2W.inverse();
    return Rt;
}

torch::Tensor getProjectionMatrix(double znear, double zfar, double fovX, double fovY) {
    double tanHalfFovY = std::tan((fovY / 2));
    double tanHalfFovX = std::tan((fovX / 2));

    double top = tanHalfFovY * znear;
    double bottom = -top;
    double right = tanHalfFovX * znear;
    double left = -right;

    Eigen::Matrix4d P = Eigen::Matrix4d::Zero();

    double z_sign = 1.0;

    P(0, 0) = 2.0 * znear / (right - left);
    P(1, 1) = 2.0 * znear / (top - bottom);
    P(0, 2) = (right + left) / (right - left);
    P(1, 2) = (top + bottom) / (top - bottom);
    P(3, 2) = z_sign;
    P(2, 2) = z_sign * zfar / (zfar - znear);
    P(2, 3) = -(zfar * znear) / (zfar - znear);

    // create torch::Tensor from Eigen::Matrix
    auto PTensor = torch::from_blob(P.data(), {4, 4}, torch::kDouble);

    // clone the tensor to allocate new memory
    return PTensor.clone();
}

double fov2focal(double fov, double pixels) {
    return pixels / (2 * std::tan(fov / 2));
}

double focal2fov(double focal, double pixels) {
    return 2 * std::atan(pixels / (2 * focal));
}

Eigen::Matrix3d qvec2rotmat(const Eigen::Quaterniond& q) {
    Eigen::Vector4d qvec = q.coeffs(); // [x, y, z, w]

    Eigen::Matrix3d rotmat;
    rotmat << 1 - 2 * qvec[2] * qvec[2] - 2 * qvec[3] * qvec[3],
        2 * qvec[1] * qvec[2] - 2 * qvec[0] * qvec[3],
        2 * qvec[3] * qvec[1] + 2 * qvec[0] * qvec[2],
        2 * qvec[1] * qvec[2] + 2 * qvec[0] * qvec[3],
        1 - 2 * qvec[1] * qvec[1] - 2 * qvec[3] * qvec[3],
        2 * qvec[2] * qvec[3] - 2 * qvec[0] * qvec[1],
        2 * qvec[3] * qvec[1] - 2 * qvec[0] * qvec[2],
        2 * qvec[2] * qvec[3] + 2 * qvec[0] * qvec[1],
        1 - 2 * qvec[1] * qvec[1] - 2 * qvec[2] * qvec[2];

    return rotmat;
}

Eigen::Quaterniond rotmat2qvec(const Eigen::Matrix3d& R) {
    Eigen::Quaterniond qvec(R);
    // the order of coefficients is different in python implementation.
    // Might be a bug here if data comes in wrong order! TODO: check
    if (qvec.w() < 0) {
        qvec.coeffs() *= -1;
    }
    return qvec;
}

unsigned char* read_image(std::filesystem::path image_path, int width, int height, int channels) {
    unsigned char* img = stbi_load(image_path.string().c_str(), &width, &height, &channels, 0);
    if (img == nullptr) {
        throw std::runtime_error("Could not load image: " + image_path.string());
    }

    return img;
}

void free_image(unsigned char* image) {
    stbi_image_free(image);
    image = nullptr;
}
